#include "hipblas.h"

#include "op_seq.h"
#include "dg_blas_calls.h"

inline void cublas_init_grid(hipblasHandle_t handle, const int numCells,
                        const double *nodeX, const double *nodeY,
                        double *x_d, double *y_d, double *xr_d, double *xs_d,
                        double *yr_d, double *ys_d) {
  double *temp_d;
  hipMalloc((void**)&temp_d, numCells * 15 * sizeof(double));

  for(int c = 0; c < numCells; c++) {
    // Get nodes for this cell (on host)
    const double n0[] = {nodeX[c * 3], nodeY[3 * c]};
    const double n1[] = {nodeX[c * 3 + 1], nodeY[3 * c + 1]};
    const double n2[] = {nodeX[c * 3 + 2], nodeY[3 * c + 2]};

    double *temp = temp_d + c * 15;
    double *x = x_d + c * 15;
    double *y = y_d + c * 15;

    double alpha = 1.0;
    hipblasDcopy(handle, 15, constants->ones_d, 1, x, 1);
    hipblasDaxpy(handle, 15, &alpha, constants->r_d, 1, x, 1);
    alpha = 0.5 * n1[0];
    hipblasDscal(handle, 15, &alpha, x, 1);
    hipblasDcopy(handle, 15, constants->ones_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, constants->s_d, 1, temp, 1);
    alpha = 0.5 * n2[0];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, x, 1);
    hipblasDcopy(handle, 15, constants->s_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, constants->r_d, 1, temp, 1);
    alpha = -0.5 * n0[0];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, x, 1);

    hipblasDcopy(handle, 15, constants->ones_d, 1, y, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, constants->r_d, 1, y, 1);
    alpha = 0.5 * n1[1];
    hipblasDscal(handle, 15, &alpha, y, 1);
    hipblasDcopy(handle, 15, constants->ones_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, constants->s_d, 1, temp, 1);
    alpha = 0.5 * n2[1];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, y, 1);
    hipblasDcopy(handle, 15, constants->s_d, 1, temp, 1);
    alpha = 1.0;
    hipblasDaxpy(handle, 15, &alpha, constants->r_d, 1, temp, 1);
    alpha = -0.5 * n0[1];
    hipblasDaxpy(handle, 15, &alpha, temp, 1, y, 1);
  }

  // HIPBLAS_OP_T because cublas is column major but constants are stored row major
  double alpha2 = 1.0;
  double beta = 0.0;
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->Dr_d, 15, x_d, 15, &beta, xr_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->Ds_d, 15, x_d, 15, &beta, xs_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->Dr_d, 15, y_d, 15, &beta, yr_d, 15);
  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 15, numCells, 15, &alpha2, constants->Ds_d, 15, y_d, 15, &beta, ys_d, 15);

  hipFree(temp_d);
}

void init_grid_blas(DGMesh *mesh) {
  // Make sure OP2 data is in the right place
  op_arg init_grid_args[] = {
    op_arg_dat(mesh->nodeX, -1, OP_ID, 3, "double", OP_READ),
    op_arg_dat(mesh->nodeY, -1, OP_ID, 3, "double", OP_READ),
    op_arg_dat(mesh->x, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(mesh->y, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(mesh->rx, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(mesh->sx, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(mesh->ry, -1, OP_ID, 15, "double", OP_WRITE),
    op_arg_dat(mesh->sy, -1, OP_ID, 15, "double", OP_WRITE)
  };
  op_mpi_halo_exchanges_cuda(mesh->cells, 8, init_grid_args);

  int setSize = mesh->x->set->size;
  double *tempX = (double *)malloc(setSize * 3 * sizeof(double));
  double *tempY = (double *)malloc(setSize * 3 * sizeof(double));
  hipMemcpy(tempX, mesh->nodeX->data_d, setSize * 3 * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(tempY, mesh->nodeY->data_d, setSize * 3 * sizeof(double), hipMemcpyDeviceToHost);

  cublas_init_grid(constants->handle, setSize, tempX, tempY, (double *)mesh->x->data_d,
                   (double *)mesh->y->data_d, (double *)mesh->rx->data_d,
                   (double *)mesh->sx->data_d, (double *)mesh->ry->data_d,
                   (double *)mesh->sy->data_d);

  free(tempX);
  free(tempY);

  // Set correct dirty bits for OP2
  op_mpi_set_dirtybit_cuda(8, init_grid_args);
}
