#include "hip/hip_runtime.h"
#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
#include "op_cuda_reduction.h"

#include "dg_compiler_defs.h"

__device__ void _p_multigrid_relaxation_chebyshev_2_gpu(const int node, const double *b,
                                               const double *diag, double *res) {
  res[node] -= b[node] / diag[node];
}

// CUDA kernel function
template<int p>
__global__ void _op_cuda_p_multigrid_relaxation_chebyshev_2(
  const int *__restrict arg0,
  const double *__restrict arg1,
  const double *__restrict arg2,
  double *arg3,
  int   set_size ) {

  const int np = (p + 1) * (p + 2) * (p + 3) / 6;

  //process set elements
  for(int n = threadIdx.x + blockIdx.x * blockDim.x;
      n < set_size * np; n += blockDim.x * gridDim.x){

    const int node = n % np;
    const int cell = n / np;
    _p_multigrid_relaxation_chebyshev_2_gpu(node,
                                       arg1+cell*DG_NP,
                                       arg2+cell*DG_NP,
                                       arg3+cell*DG_NP);
  }
}


//host stub function
void custom_kernel_p_multigrid_relaxation_chebyshev_2(const int order, char const *name, op_set set,
  op_arg arg0,
  op_arg arg1,
  op_arg arg2,
  op_arg arg3){

  int nargs = 4;
  op_arg args[4];

  args[0] = arg0;
  args[1] = arg1;
  args[2] = arg2;
  args[3] = arg3;

  if (OP_diags>2) {
    printf(" kernel routine w/o indirection:  p_multigrid_relaxation_chebyshev_2");
  }

  int set_size = op_mpi_halo_exchanges_grouped(set, nargs, args, 2, 0);
  if (set_size > 0) {

    //set CUDA execution parameters
    const int nthread = 256;
    const int nblocks = 200 < (set->size * DG_NP) / nthread + 1 ? 200 : (set->size * DG_NP) / nthread + 1;

    switch(order) {
      case 1:
        _op_cuda_p_multigrid_relaxation_chebyshev_2<1><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          set->size );
        break;
      case 2:
        _op_cuda_p_multigrid_relaxation_chebyshev_2<2><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          set->size );
        break;
      case 3:
        _op_cuda_p_multigrid_relaxation_chebyshev_2<3><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          set->size );
        break;
      case 4:
        _op_cuda_p_multigrid_relaxation_chebyshev_2<4><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          set->size );
        break;
      case 5:
        _op_cuda_p_multigrid_relaxation_chebyshev_2<5><<<nblocks,nthread>>>(
          (int *) arg0.data_d,
          (double *) arg1.data_d,
          (double *) arg2.data_d,
          (double *) arg3.data_d,
          set->size );
        break;
    }
  }
  op_mpi_set_dirtybit_cuda(nargs, args);
  cutilSafeCall(hipDeviceSynchronize());
}
