#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
#include "op_cuda_reduction.h"

#include "dg_compiler_defs.h"

#include "kernels/non_templated_soa.h"
#include "kernels/templated_soa.h"

void custom_kernel_gemv(op_set set, const bool t, const int m, const int n, const DG_FP alpha,
  const DG_FP beta, const DG_FP *matrix, op_dat x, op_dat y) {

  int nargs = 2;
  op_arg args[2] = {
    op_arg_dat(x, -1, OP_ID, x->dim, DG_FP_STR, OP_READ),
    op_arg_dat(y, -1, OP_ID, y->dim, DG_FP_STR, OP_WRITE)
  };

  int set_size = op_mpi_halo_exchanges_grouped(set, nargs, args, 2);
  if (set_size > 0) {
    DG_FP *matrix_d;
    hipMalloc(&matrix_d, m * n * sizeof(DG_FP));
    hipMemcpy(matrix_d, matrix, m * n * sizeof(DG_FP), hipMemcpyHostToDevice);

    //set CUDA execution parameters
    int nthread = 256;
    const int nblocks = set->size / nthread + 1;
    const int strideX = getSetSizeFromOpArg(&args[0]);
    const int strideY = getSetSizeFromOpArg(&args[1]);

    if(t) {
      switch(m) {
        // The number of nodes for each order
        case 4:
          templated_cuda_gemm_T_gpu<4><<<nblocks,nthread,m*n*sizeof(double)>>>(n,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        case 10:
          templated_cuda_gemm_T_gpu<10><<<nblocks,nthread,m*n*sizeof(double)>>>(n,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        case 20:
          templated_cuda_gemm_T_gpu<20><<<nblocks,nthread,m*n*sizeof(double)>>>(n,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        // The number of face nodes for each order
        case 12:
          templated_cuda_gemm_T_gpu<12><<<nblocks,nthread,m*n*sizeof(double)>>>(n,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        case 24:
          templated_cuda_gemm_T_gpu<24><<<nblocks,nthread,m*n*sizeof(double)>>>(n,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        case 40:
          templated_cuda_gemm_T_gpu<40><<<nblocks,nthread,m*n*sizeof(double)>>>(n,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        default:
          cuda_gemm_T_gpu<<<nblocks,nthread,m*n*sizeof(double)>>>(m, n, strideX, strideY, alpha, beta,
                                               matrix_d, (double *) args[0].data_d,
                                               (double *) args[1].data_d, set->size);
      }
    } else {
      switch(n) {
        // The number of nodes for each order
        case 4:
          templated_cuda_gemm_gpu<4><<<nblocks,nthread,m*n*sizeof(double)>>>(m,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        case 10:
          templated_cuda_gemm_gpu<10><<<nblocks,nthread,m*n*sizeof(double)>>>(m,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        case 20:
          templated_cuda_gemm_gpu<20><<<nblocks,nthread,m*n*sizeof(double)>>>(m,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        // The number of face nodes for each order
        case 12:
          templated_cuda_gemm_gpu<12><<<nblocks,nthread,m*n*sizeof(double)>>>(m,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        case 24:
          templated_cuda_gemm_gpu<24><<<nblocks,nthread,m*n*sizeof(double)>>>(m,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        case 40:
          templated_cuda_gemm_gpu<40><<<nblocks,nthread,m*n*sizeof(double)>>>(m,
                                              strideX, strideY, alpha, beta,
                                              matrix_d, (double *) args[0].data_d,
                                              (double *) args[1].data_d, set->size);
          break;
        default:
          cuda_gemm_gpu<<<nblocks,nthread,m*n*sizeof(double)>>>(m, n, strideX, strideY, alpha, beta,
                                             matrix_d, (double *) args[0].data_d,
                                             (double *) args[1].data_d, set->size);
      }
    }

    hipFree(matrix_d);
  }
  op_mpi_set_dirtybit_cuda(nargs, args);
  cutilSafeCall(hipDeviceSynchronize());
}
