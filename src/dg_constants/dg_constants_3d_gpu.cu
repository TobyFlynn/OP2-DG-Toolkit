#include "dg_constants/dg_constants_3d.h"

#include "op_cuda_rt_support.h"

#include <stdexcept>

__constant__ DG_FP *dg_r_kernel;
__constant__ DG_FP *dg_s_kernel;
__constant__ DG_FP *dg_t_kernel;
__constant__ DG_FP *dg_Dr_kernel;
__constant__ DG_FP *dg_Ds_kernel;
__constant__ DG_FP *dg_Dt_kernel;
__constant__ DG_FP *dg_Drw_kernel;
__constant__ DG_FP *dg_Dsw_kernel;
__constant__ DG_FP *dg_Dtw_kernel;
__constant__ DG_FP *dg_Mass_kernel;
__constant__ DG_FP *dg_InvMass_kernel;
__constant__ DG_FP *dg_InvV_kernel;
__constant__ DG_FP *dg_V_kernel;
__constant__ DG_FP *dg_Lift_kernel;
__constant__ DG_FP *dg_MM_F0_kernel;
__constant__ DG_FP *dg_MM_F1_kernel;
__constant__ DG_FP *dg_MM_F2_kernel;
__constant__ DG_FP *dg_MM_F3_kernel;
__constant__ DG_FP *dg_Emat_kernel;
__constant__ DG_FP *dg_Interp_kernel;
__constant__ DG_FP *dg_cub3d_Interp_kernel;
__constant__ DG_FP *dg_cub3d_Proj_kernel;
__constant__ DG_FP *dg_cub3d_PDr_kernel;
__constant__ DG_FP *dg_cub3d_PDs_kernel;
__constant__ DG_FP *dg_cub3d_PDt_kernel;

DG_FP *dg_r_d;
DG_FP *dg_s_d;
DG_FP *dg_t_d;
DG_FP *dg_Dr_d;
DG_FP *dg_Ds_d;
DG_FP *dg_Dt_d;
DG_FP *dg_Drw_d;
DG_FP *dg_Dsw_d;
DG_FP *dg_Dtw_d;
DG_FP *dg_Mass_d;
DG_FP *dg_InvMass_d;
DG_FP *dg_InvV_d;
DG_FP *dg_V_d;
DG_FP *dg_Lift_d;
DG_FP *dg_MM_F0_d;
DG_FP *dg_MM_F1_d;
DG_FP *dg_MM_F2_d;
DG_FP *dg_MM_F3_d;
DG_FP *dg_Emat_d;
DG_FP *dg_Interp_d;
DG_FP *dg_cub3d_Interp_d;
DG_FP *dg_cub3d_Proj_d;
DG_FP *dg_cub3d_PDr_d;
DG_FP *dg_cub3d_PDs_d;
DG_FP *dg_cub3d_PDt_d;

float *dg_Dr_sp_d;
float *dg_Ds_sp_d;
float *dg_Dt_sp_d;
float *dg_Drw_sp_d;
float *dg_Dsw_sp_d;
float *dg_Dtw_sp_d;
float *dg_Mass_sp_d;
float *dg_InvMass_sp_d;
float *dg_InvV_sp_d;
float *dg_V_sp_d;
float *dg_Lift_sp_d;
float *dg_Emat_sp_d;
float *dg_Interp_sp_d;

void DGConstants3D::transfer_kernel_ptrs() {
  // Allocate device memory
  cutilSafeCall(hipMalloc(&dg_r_d, N_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_s_d, N_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_t_d, N_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Dr_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Ds_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Dt_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Drw_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Dsw_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Dtw_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Mass_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_InvMass_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_InvV_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_V_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Lift_d, N_max * DG_NUM_FACES * Nfp_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_MM_F0_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_MM_F1_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_MM_F2_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_MM_F3_d, N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Emat_d, N_max * DG_NUM_FACES * Nfp_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_Interp_d, N_max * N_max * Np_max * Np_max * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_cub3d_Interp_d, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_cub3d_Proj_d, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_cub3d_PDr_d, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_cub3d_PDs_d, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP)));
  cutilSafeCall(hipMalloc(&dg_cub3d_PDt_d, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP)));

  // Transfer matrices to device
  cutilSafeCall(hipMemcpy(dg_r_d, r_ptr, N_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_s_d, s_ptr, N_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_t_d, t_ptr, N_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Dr_d, Dr_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Ds_d, Ds_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Dt_d, Dt_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Drw_d, Drw_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Dsw_d, Dsw_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Dtw_d, Dtw_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Mass_d, mass_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_InvMass_d, invMass_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_InvV_d, invV_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_V_d, v_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Lift_d, lift_ptr, N_max * DG_NUM_FACES * Nfp_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_MM_F0_d, mmF0_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_MM_F1_d, mmF1_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_MM_F2_d, mmF2_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_MM_F3_d, mmF3_ptr, N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Emat_d, eMat_ptr, N_max * DG_NUM_FACES * Nfp_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Interp_d, order_interp_ptr, N_max * N_max * Np_max * Np_max * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_cub3d_Interp_d, cubInterp_ptr, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_cub3d_Proj_d, cubProj_ptr, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_cub3d_PDr_d, cubPDr_ptr, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_cub3d_PDs_d, cubPDs_ptr, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_cub3d_PDt_d, cubPDt_ptr, DG_NP * DG_CUB_3D_NP * sizeof(DG_FP), hipMemcpyHostToDevice));

  // Set up pointers that are accessible from the device
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_r_kernel), &dg_r_d, sizeof(dg_r_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_s_kernel), &dg_s_d, sizeof(dg_s_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_t_kernel), &dg_t_d, sizeof(dg_t_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Dr_kernel), &dg_Dr_d, sizeof(dg_Dr_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Ds_kernel), &dg_Ds_d, sizeof(dg_Ds_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Dt_kernel), &dg_Dt_d, sizeof(dg_Dt_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Drw_kernel), &dg_Drw_d, sizeof(dg_Drw_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Dsw_kernel), &dg_Dsw_d, sizeof(dg_Dsw_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Dtw_kernel), &dg_Dtw_d, sizeof(dg_Dtw_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Mass_kernel), &dg_Mass_d, sizeof(dg_Mass_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_InvMass_kernel), &dg_InvMass_d, sizeof(dg_InvMass_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_InvV_kernel), &dg_InvV_d, sizeof(dg_InvV_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_V_kernel), &dg_V_d, sizeof(dg_V_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Lift_kernel), &dg_Lift_d, sizeof(dg_Lift_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_MM_F0_kernel), &dg_MM_F0_d, sizeof(dg_MM_F0_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_MM_F1_kernel), &dg_MM_F1_d, sizeof(dg_MM_F1_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_MM_F2_kernel), &dg_MM_F2_d, sizeof(dg_MM_F2_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_MM_F3_kernel), &dg_MM_F3_d, sizeof(dg_MM_F3_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Emat_kernel), &dg_Emat_d, sizeof(dg_Emat_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_Interp_kernel), &dg_Interp_d, sizeof(dg_Interp_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_cub3d_Interp_kernel), &dg_cub3d_Interp_d, sizeof(dg_cub3d_Interp_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_cub3d_Proj_kernel), &dg_cub3d_Proj_d, sizeof(dg_cub3d_Proj_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_cub3d_PDr_kernel), &dg_cub3d_PDr_d, sizeof(dg_cub3d_Interp_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_cub3d_PDs_kernel), &dg_cub3d_PDs_d, sizeof(dg_cub3d_PDs_d)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dg_cub3d_PDt_kernel), &dg_cub3d_PDt_d, sizeof(dg_cub3d_PDt_d)));

  cutilSafeCall(hipMalloc(&dg_Dr_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Ds_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Dt_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Drw_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Dsw_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Dtw_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Mass_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_InvMass_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_InvV_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_V_sp_d, N_max * Np_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Lift_sp_d, N_max * DG_NUM_FACES * Nfp_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Emat_sp_d, N_max * DG_NUM_FACES * Nfp_max * Np_max * sizeof(float)));
  cutilSafeCall(hipMalloc(&dg_Interp_sp_d, N_max * N_max * Np_max * Np_max * sizeof(float)));

  cutilSafeCall(hipMemcpy(dg_Dr_sp_d, Dr_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Ds_sp_d, Ds_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Dt_sp_d, Dt_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Drw_sp_d, Drw_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Dsw_sp_d, Dsw_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Dtw_sp_d, Dtw_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Mass_sp_d, mass_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_InvMass_sp_d, invMass_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_InvV_sp_d, invV_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_V_sp_d, v_ptr_sp, N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Lift_sp_d, lift_ptr_sp, N_max * DG_NUM_FACES * Nfp_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Emat_sp_d, eMat_ptr_sp, N_max * DG_NUM_FACES * Nfp_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(dg_Interp_sp_d, order_interp_ptr_sp, N_max * N_max * Np_max * Np_max * sizeof(float), hipMemcpyHostToDevice));
}

void DGConstants3D::clean_up_kernel_ptrs() {
  hipFree(dg_r_d);
  hipFree(dg_s_d);
  hipFree(dg_t_d);
  hipFree(dg_Dr_d);
  hipFree(dg_Ds_d);
  hipFree(dg_Dt_d);
  hipFree(dg_Drw_d);
  hipFree(dg_Dsw_d);
  hipFree(dg_Dtw_d);
  hipFree(dg_Mass_d);
  hipFree(dg_InvMass_d);
  hipFree(dg_InvV_d);
  hipFree(dg_V_d);
  hipFree(dg_Lift_d);
  hipFree(dg_MM_F0_d);
  hipFree(dg_MM_F1_d);
  hipFree(dg_MM_F2_d);
  hipFree(dg_MM_F3_d);
  hipFree(dg_Emat_d);
  hipFree(dg_Interp_d);

  hipFree(dg_Dr_sp_d);
  hipFree(dg_Ds_sp_d);
  hipFree(dg_Dt_sp_d);
  hipFree(dg_Drw_sp_d);
  hipFree(dg_Dsw_sp_d);
  hipFree(dg_Dtw_sp_d);
  hipFree(dg_Mass_sp_d);
  hipFree(dg_InvMass_sp_d);
  hipFree(dg_InvV_sp_d);
  hipFree(dg_V_sp_d);
  hipFree(dg_Lift_sp_d);
  hipFree(dg_Emat_sp_d);
  hipFree(dg_Interp_sp_d);
}

DG_FP* DGConstants3D::get_mat_ptr_kernel(Constant_Matrix matrix) {
  switch(matrix) {
    case R:
      return dg_r_d;
    case S:
      return dg_s_d;
    case T:
      return dg_t_d;
    case DR:
      return dg_Dr_d;
    case DS:
      return dg_Ds_d;
    case DT:
      return dg_Dt_d;
    case DRW:
      return dg_Drw_d;
    case DSW:
      return dg_Dsw_d;
    case DTW:
      return dg_Dtw_d;
    case MASS:
      return dg_Mass_d;
    case INV_MASS:
      return dg_InvMass_d;
    case INV_V:
      return dg_InvV_d;
    case V:
      return dg_V_d;
    case LIFT:
      return dg_Lift_d;
    case MM_F0:
      return dg_MM_F0_d;
    case MM_F1:
      return dg_MM_F1_d;
    case MM_F2:
      return dg_MM_F2_d;
    case MM_F3:
      return dg_MM_F3_d;
    case EMAT:
      return dg_Emat_d;
    case INTERP_MATRIX_ARRAY:
      return dg_Interp_d;
    case CUB3D_INTERP:
      return dg_cub3d_Interp_d;
    case CUB3D_PROJ:
      return dg_cub3d_Proj_d;
    case CUB3D_PDR:
      return dg_cub3d_PDr_d;
    case CUB3D_PDS:
      return dg_cub3d_PDs_d;
    case CUB3D_PDT:
      return dg_cub3d_PDt_d;
    default:
      throw std::runtime_error("This constant matrix is not supported by DGConstants3D\n");
      return nullptr;
  }
}

float* DGConstants3D::get_mat_ptr_kernel_sp(Constant_Matrix matrix) {
  switch(matrix) {
    case DR:
      return dg_Dr_sp_d;
    case DS:
      return dg_Ds_sp_d;
    case DT:
      return dg_Dt_sp_d;
    case DRW:
      return dg_Drw_sp_d;
    case DSW:
      return dg_Dsw_sp_d;
    case DTW:
      return dg_Dtw_sp_d;
    case MASS:
      return dg_Mass_sp_d;
    case INV_MASS:
      return dg_InvMass_sp_d;
    case INV_V:
      return dg_InvV_sp_d;
    case V:
      return dg_V_sp_d;
    case LIFT:
      return dg_Lift_sp_d;
    case EMAT:
      return dg_Emat_sp_d;
    case INTERP_MATRIX_ARRAY:
      return dg_Interp_sp_d;
    default:
      throw std::runtime_error("This sp constant matrix is not supported by DGConstants3D\n");
      return nullptr;
  }
}
