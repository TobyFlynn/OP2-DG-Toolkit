#include "dg_constants.h"

#include "dg_global_constants.h"

DGConstants::DGConstants() {
  // Cubature constants
  hipMalloc((void**)&cubDr_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubDr_d, cubDr_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubDs_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubDs_d, cubDs_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubV_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubV_d, cubV_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubVDr_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubVDr_d, cubVDr_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubVDs_d, 46 * 15 * sizeof(double));
  hipMemcpy(cubVDs_d, cubVDs_g, 46 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubW_d, 46 * sizeof(double));
  hipMemcpy(cubW_d, cubW_d, 46 * sizeof(double), hipMemcpyHostToDevice);
  // Grad constants
  hipMalloc((void**)&Dr_d, 15 * 15 * sizeof(double));
  hipMemcpy(Dr_d, Dr_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Drw_d, 15 * 15 * sizeof(double));
  hipMemcpy(Drw_d, Drw_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Ds_d, 15 * 15 * sizeof(double));
  hipMemcpy(Ds_d, Ds_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Dsw_d, 15 * 15 * sizeof(double));
  hipMemcpy(Dsw_d, Dsw_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  // Gauss constants
  hipMalloc((void**)&gaussW_d, 7 * sizeof(double));
  hipMemcpy(gaussW_d, gaussW_g, 7 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF0Dr_d, gF0Dr_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0DrR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF0DrR_d, gF0DrR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF0Ds_d, gF0Ds_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0DsR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF0DsR_d, gF0DsR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF1Dr_d, gF1Dr_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1DrR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF1DrR_d, gF1DrR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF1Ds_d, gF1Ds_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1DsR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF1DsR_d, gF1DsR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2Dr_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF2Dr_d, gF2Dr_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2DrR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF2DrR_d, gF2DrR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2Ds_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF2Ds_d, gF2Ds_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2DsR_d, 7 * 15 * sizeof(double));
  hipMemcpy(gF2DsR_d, gF2DsR_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp0_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp0_d, gFInterp0_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp0R_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp0R_d, gFInterp0R_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp1_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp1_d, gFInterp1_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp1R_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp1R_d, gFInterp1R_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp2_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp2_d, gFInterp2_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp2R_d, 7 * 15 * sizeof(double));
  hipMemcpy(gFInterp2R_d, gFInterp2R_g, 7 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gInterp_d, 21 * 15 * sizeof(double));
  hipMemcpy(gInterp_d, gInterp_g, 21 * 15 * sizeof(double), hipMemcpyHostToDevice);
  // Other constants
  hipMalloc((void**)&invMass_d, 15 * 15 * sizeof(double));
  hipMemcpy(invMass_d, invMass_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&lift_d, 15 * 15 * sizeof(double));
  hipMemcpy(lift_d, lift_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&mass_d, 15 * 15 * sizeof(double));
  hipMemcpy(mass_d, mass_g, 15 * 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&r_d, 15 * sizeof(double));
  hipMemcpy(r_d, r_g, 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&s_d, 15 * sizeof(double));
  hipMemcpy(s_d, s_g, 15 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&ones_d, 15 * sizeof(double));
  hipMemcpy(ones_d, ones_g, 15 * sizeof(double), hipMemcpyHostToDevice);

  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
}

DGConstants::~DGConstants() {
  // Cubature constants
  hipFree(cubDr_d);
  hipFree(cubDs_d);
  hipFree(cubV_d);
  hipFree(cubVDr_d);
  hipFree(cubVDs_d);
  hipFree(cubW_d);
  // Grad constants
  hipFree(Dr_d);
  hipFree(Drw_d);
  hipFree(Ds_d);
  hipFree(Dsw_d);
  // Gauss constants
  hipFree(gaussW_d);
  hipFree(gF0Dr_d);
  hipFree(gF0DrR_d);
  hipFree(gF0Ds_d);
  hipFree(gF0DsR_d);
  hipFree(gF1Dr_d);
  hipFree(gF1DrR_d);
  hipFree(gF1Ds_d);
  hipFree(gF1DsR_d);
  hipFree(gF2Dr_d);
  hipFree(gF2DrR_d);
  hipFree(gF2Ds_d);
  hipFree(gF2DsR_d);
  hipFree(gFInterp0_d);
  hipFree(gFInterp0R_d);
  hipFree(gFInterp1_d);
  hipFree(gFInterp1R_d);
  hipFree(gFInterp2_d);
  hipFree(gFInterp2R_d);
  hipFree(gInterp_d);
  // Other constants
  hipFree(invMass_d);
  hipFree(lift_d);
  hipFree(mass_d);
  hipFree(r_d);
  hipFree(s_d);
  hipFree(ones_d);

  hipblasDestroy(handle);
}

double* DGConstants::get_ptr(Constant_Matrix mat) {
  switch(mat) {
    case CUB_DR:
      return cubDr_d;
    case CUB_DS:
      return cubDs_d;
    case CUB_V:
      return cubV_d;
    case CUB_VDR:
      return cubVDr_d;
    case CUB_VDS:
      return cubVDs_d;
    case CUB_W:
      return cubW_d;
    case DR:
      return Dr_d;
    case DRW:
      return Drw_d;
    case DS:
      return Ds_d;
    case DSW:
      return Dsw_d;
    case GAUSS_W:
      return gaussW_d;
    case GAUSS_F0DR:
      return gF0Dr_d;
    case GAUSS_F0DR_R:
      return gF0DrR_d;
    case GAUSS_F0DS:
      return gF0Ds_d;
    case GAUSS_F0DS_R:
      return gF0DsR_d;
    case GAUSS_F1DR:
      return gF1Dr_d;
    case GAUSS_F1DR_R:
      return gF1DrR_d;
    case GAUSS_F1DS:
      return gF1Ds_d;
    case GAUSS_F1DS_R:
      return gF1DsR_d;
    case GAUSS_F2DR:
      return gF2Dr_d;
    case GAUSS_F2DR_R:
      return gF2DrR_d;
    case GAUSS_F2DS:
      return gF2Ds_d;
    case GAUSS_F2DS_R:
      return gF2DsR_d;
    case GAUSS_FINTERP0:
      return gFInterp0_d;
    case GAUSS_FINTERP0_R:
      return gFInterp0R_d;
    case GAUSS_FINTERP1:
      return gFInterp1_d;
    case GAUSS_FINTERP1_R:
      return gFInterp1R_d;
    case GAUSS_FINTERP2:
      return gFInterp2_d;
    case GAUSS_FINTERP2_R:
      return gFInterp2R_d;
    case GAUSS_INTERP:
      return gInterp_d;
    case INV_MASS:
      return invMass_d;
    case LIFT:
      return lift_d;
    case MASS:
      return mass_d;
    case R:
      return r_d;
    case S:
      return s_d;
    case ONES:
      return ones_d;
  }
  return nullptr;
}
