#include "dg_constants.h"

#include "dg_global_constants.h"

DGConstants::DGConstants(const int n) {
  setup(n);

  // Cubature constants
  hipMalloc((void**)&cubDr_d, DG_CUB_NP * DG_NP * sizeof(double));
  hipMemcpy(cubDr_d, &cubDr_g[(N - 1) * DG_CUB_NP * DG_NP], DG_CUB_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubDs_d, DG_CUB_NP * DG_NP * sizeof(double));
  hipMemcpy(cubDs_d, &cubDs_g[(N - 1) * DG_CUB_NP * DG_NP], DG_CUB_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubV_d, DG_CUB_NP * DG_NP * sizeof(double));
  hipMemcpy(cubV_d, &cubV_g[(N - 1) * DG_CUB_NP * DG_NP], DG_CUB_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubVDr_d, DG_CUB_NP * DG_NP * sizeof(double));
  hipMemcpy(cubVDr_d, &cubVDr_g[(N - 1) * DG_CUB_NP * DG_NP], DG_CUB_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubVDs_d, DG_CUB_NP * DG_NP * sizeof(double));
  hipMemcpy(cubVDs_d, &cubVDs_g[(N - 1) * DG_CUB_NP * DG_NP], DG_CUB_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&cubW_d, DG_CUB_NP * sizeof(double));
  hipMemcpy(cubW_d, &cubW_g[(N - 1) * DG_CUB_NP], DG_CUB_NP * sizeof(double), hipMemcpyHostToDevice);
  // Grad constants
  hipMalloc((void**)&Dr_d, DG_NP * DG_NP * sizeof(double));
  hipMemcpy(Dr_d, &Dr_g[(N - 1) * DG_NP * DG_NP], DG_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Drw_d, DG_NP * DG_NP * sizeof(double));
  hipMemcpy(Drw_d, &Drw_g[(N - 1) * DG_NP * DG_NP], DG_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Ds_d, DG_NP * DG_NP * sizeof(double));
  hipMemcpy(Ds_d, &Ds_g[(N - 1) * DG_NP * DG_NP], DG_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&Dsw_d, DG_NP * DG_NP * sizeof(double));
  hipMemcpy(Dsw_d, &Dsw_g[(N - 1) * DG_NP * DG_NP], DG_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  // Gauss constants
  hipMalloc((void**)&gaussW_d, DG_GF_NP * sizeof(double));
  hipMemcpy(gaussW_d, &gaussW_g[(N - 1) * DG_GF_NP], DG_GF_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0Dr_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gF0Dr_d, &gF0Dr_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF0Ds_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gF0Ds_d, &gF0Ds_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1Dr_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gF1Dr_d, &gF1Dr_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF1Ds_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gF1Ds_d, &gF1Ds_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2Dr_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gF2Dr_d, &gF2Dr_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gF2Ds_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gF2Ds_d, &gF2Ds_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp0_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gFInterp0_d, &gFInterp0_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp1_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gFInterp1_d, &gFInterp1_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gFInterp2_d, DG_GF_NP * DG_NP * sizeof(double));
  hipMemcpy(gFInterp2_d, &gFInterp2_g[(N - 1) * DG_GF_NP * DG_NP], DG_GF_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&gInterp_d, DG_G_NP * DG_NP * sizeof(double));
  hipMemcpy(gInterp_d, &gInterp_g[(N - 1) * DG_G_NP * DG_NP], DG_G_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  // Other constants
  hipMalloc((void**)&invMass_d, DG_NP * DG_NP * sizeof(double));
  hipMemcpy(invMass_d, &invMass_g[(N - 1) * DG_NP * DG_NP], DG_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&lift_d, DG_NP * DG_NPF * 3 * sizeof(double));
  hipMemcpy(lift_d, &lift_g[(N - 1) * DG_NP * 3 * DG_NPF], DG_NP * DG_NPF * 3 * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&mass_d, DG_NP * DG_NP * sizeof(double));
  hipMemcpy(mass_d, &mass_g[(N - 1) * DG_NP * DG_NP], DG_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&v_d, DG_NP * DG_NP * sizeof(double));
  hipMemcpy(v_d, &v_g[(N - 1) * DG_NP * DG_NP], DG_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&invV_d, DG_NP * DG_NP * sizeof(double));
  hipMemcpy(invV_d, &invV_g[(N - 1) * DG_NP * DG_NP], DG_NP * DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&r_d, DG_NP * sizeof(double));
  hipMemcpy(r_d, &r_g[(N - 1) * DG_NP], DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&s_d, DG_NP * sizeof(double));
  hipMemcpy(s_d, &s_g[(N - 1) * DG_NP], DG_NP * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**)&ones_d, DG_NP * sizeof(double));
  hipMemcpy(ones_d, &ones_g[(N - 1) * DG_NP], DG_NP * sizeof(double), hipMemcpyHostToDevice);
}

DGConstants::~DGConstants() {
  // Cubature constants
  hipFree(cubDr_d);
  hipFree(cubDs_d);
  hipFree(cubV_d);
  hipFree(cubVDr_d);
  hipFree(cubVDs_d);
  hipFree(cubW_d);
  // Grad constants
  hipFree(Dr_d);
  hipFree(Drw_d);
  hipFree(Ds_d);
  hipFree(Dsw_d);
  // Gauss constants
  hipFree(gaussW_d);
  hipFree(gF0Dr_d);
  hipFree(gF0Ds_d);
  hipFree(gF1Dr_d);
  hipFree(gF1Ds_d);
  hipFree(gF2Dr_d);
  hipFree(gF2Ds_d);
  hipFree(gFInterp0_d);
  hipFree(gFInterp1_d);
  hipFree(gFInterp2_d);
  hipFree(gInterp_d);
  // Other constants
  hipFree(invMass_d);
  hipFree(lift_d);
  hipFree(mass_d);
  hipFree(v_d);
  hipFree(invV_d);
  hipFree(r_d);
  hipFree(s_d);
  hipFree(ones_d);
}

double* DGConstants::get_ptr(Constant_Matrix mat) {
  switch(mat) {
    case CUB_DR:
      return cubDr_d;
    case CUB_DS:
      return cubDs_d;
    case CUB_V:
      return cubV_d;
    case CUB_VDR:
      return cubVDr_d;
    case CUB_VDS:
      return cubVDs_d;
    case CUB_W:
      return cubW_d;
    case DR:
      return Dr_d;
    case DRW:
      return Drw_d;
    case DS:
      return Ds_d;
    case DSW:
      return Dsw_d;
    case GAUSS_W:
      return gaussW_d;
    case GAUSS_F0DR:
      return gF0Dr_d;
    case GAUSS_F0DS:
      return gF0Ds_d;
    case GAUSS_F1DR:
      return gF1Dr_d;
    case GAUSS_F1DS:
      return gF1Ds_d;
    case GAUSS_F2DR:
      return gF2Dr_d;
    case GAUSS_F2DS:
      return gF2Ds_d;
    case GAUSS_FINTERP0:
      return gFInterp0_d;
    case GAUSS_FINTERP1:
      return gFInterp1_d;
    case GAUSS_FINTERP2:
      return gFInterp2_d;
    case GAUSS_INTERP:
      return gInterp_d;
    case INV_MASS:
      return invMass_d;
    case LIFT:
      return lift_d;
    case MASS:
      return mass_d;
    case V:
      return v_d;
    case INV_V:
      return invV_d;
    case R:
      return r_d;
    case S:
      return s_d;
    case ONES:
      return ones_d;
  }
  return nullptr;
}
